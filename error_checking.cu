#include "hip/hip_runtime.h"
int main(int argc, char** argv)
  {
    // ...

    unsigned char cudaFail;                                         //  Host-side flag for error testing.
    unsigned char* cudaFail_d;                                      //  Device-side flag for error testing.
    hipError_t cudaErr;

    // ...
                                                                    //  Know when device-side allocation fails.
    if(hipMalloc((void**)&data_d, data_size * sizeof(float)) != hipSuccess)
      {
        printf("CUDA ERROR: Unable to allocate device-side data.\n");
        exit(1);
      }

    // ...
                                                                    //  Know when copying to device fails.
    if(hipMemcpy(ctr_d, &ctr, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
      {
        printf("CUDA ERROR: Unable to copy counter to device.\n");
        exit(1);
      }

    // ...
                                                                    //  Set your custom error flag to 1.
                                                                    //  If all goes well device-side, it should remain 1.
    cudaFail = 1;                                                   //  (Innocent until proven guilty.)
    if(hipMemcpy(cudaFail_d, &cudaFail, sizeof(char), hipMemcpyHostToDevice) != hipSuccess)
      {
        printf("CUDA ERROR: Unable to set device-side failure-flag.\n");
        exit(1);
      }

    // ...
                                                                    //  Call kernel with your custom device-side error flag included.
    YourKernelCall<<<ceil(float(len_h) / float(BLOCK_SIZE)), BLOCK_SIZE>>>(/* Blah, blah, blah, arguments */, cudaFail_d);

    hipDeviceSynchronize();

    cudaErr = hipGetLastError();                                   //  Retrieve built-in device-side error signal.
    if(cudaErr != hipSuccess)                                      //  Something went wrong on the device. CUDA reports it here.
      {
        printf("CUDA ERROR: Kernel failed.\n");
        printf("  %s\n", hipGetErrorString(cudaErr));
        exit(1);
      }
                                                                    //  Retrieve your custom device-side error flag.
    cudaErr = hipMemcpy(&cudaFail, cudaFail_d, sizeof(char), hipMemcpyDeviceToHost);
    if(cudaErr != hipSuccess)                                      //  Did the retrieval itself fail?
      {
        printf("CUDA ERROR: Unable to copy kernel's error flag to host.\n");
        printf("  %s\n", hipGetErrorString(cudaErr));
        exit(1);
      }
    if(cudaFail == 0)                                               //  Does your custom error flag indicate failure?
      {
        printf("CUSTOM ERROR: Kernel failed.\n");
        exit(1);
      }

    // ...

    return 0;
  }

/* Your kernel. Whatever else you designed it to do, include the device-side error flag as an argument.
   If one of your routines fails without killing the whole kernel, you can write your own values here
   and test them when control returns to the host. */
__global__ void YourKernelCall(/* Blah, blah, blah, arguments */, unsigned char* cudaFail_g)
  {
    // ...

    if(something_went_wrong)                                        //  Your own test condition:
      *cudaFail_g = 0;                                              //  indicate an outcome that is undesirable, though not terminal.
                                                                    //  Find out about it host-side.
    // ...
  }